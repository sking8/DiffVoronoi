#include "hip/hip_runtime.h"
#include "VoronoiField.h"
#include "TopoCellType.h"

namespace Meso {
	//////////////////////////////////////////////////////////////////////////
	////initialization
	template<int d> void VoronoiField<d>::Initialize(const Grid<d> _grid, const Array<VectorD>& points, const int _nb_n, int _beta, real _alpha, real _c)
	{
		int p_num = (int)points.size();
		grid = _grid;
		nb_n = _nb_n;
		particles.Resize(p_num);
		particles.xRef() = points;
		beta = _beta;
		ArrayFunc::Fill(particles.DRef(), _alpha * MatrixD::Identity());
		c = _c;

		////grid attributes
		rho.Init(grid);
		rho.Fill((real)0);
		soft_max_sum.Init(grid);
		soft_max_sum.Fill((real)0);
		active.Init(grid); //Fan: need to consider this, initialize outside of the class
		active.Fill((int)TopoCellType::Active);

		////nb attributes
		int cell_num = grid.Counts().prod();
		nbs_searcher = std::make_shared<NeighborKDTree<d>>();

		nbs_c.Init(grid);
		/*int default_nb_p_num = 16;
		for (int i = 0; i < cell_num; i++)nbs_c[i].reserve(default_nb_p_num);*/

		//nbs_p.resize(p_num);
		//int default_nb_cell_num = pow(4, d);
		//for (int i = 0; i < p_num; i++)nbs_p[i].reserve(default_nb_cell_num);

		drho_dx.Init(grid);
		//drho_dD.resize(cell_num);
	}

	//////////////////////////////////////////////////////////////////////////
	////field updates

	template<int d> void VoronoiField<d>::Advance(DriverMetaData& metadata)
	{
		Update_A();
		Update_Neighbors();
		Update_Softmax_Sum();
		Update_Rho();
		//Numerical_Derivative_DRho_DX();
	}

	template<int d> void VoronoiField<d>::Update_A()
	{
		int p_num = particles.Size();
#pragma omp parallel for
		for (int i = 0; i < p_num; i++) { //Fan: to add exec particles
			particles.A(i) = particles.D(i) * particles.D(i).transpose() + epsi_A * MatrixD::Identity();
			/*Info("p: {}", i);
			std::cout << "A: " << particles.A(i) << std::endl;
			std::cout << "D: " << particles.D(i) << std::endl;*/
		}
	}

	template<int d> void VoronoiField<d>::Update_Neighbors()
	{
		////nb particles of each cell
		nbs_searcher->Update_Points(particles.xRef());

		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				nbs_c(cell).clear();
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) { return; }

				nbs_searcher->Find_K_Nearest_Nbs(pos, nb_n, nbs_c(cell));
				int nb_n = nbs_c(cell).size();
				if (nb_n > 0) {
					drho_dx(cell).resize(nb_n);
					//drho_dD[idx].resize(nb_c_n);
				}
			}
		);
	}

	template<int d> void VoronoiField<d>::Update_Softmax_Sum()
	{
		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) {
					soft_max_sum(cell) = (real)0.; return;
				}

				real sm_sum = (real)0;
				int nb_c_n = nbs_c(cell).size();
				for (int j = 0; j < nb_c_n; j++) {
					int pid = nbs_c(cell)[j];
					real dis = Dist(pid, pos);
					sm_sum += exp(-dis);
				}
				soft_max_sum(cell) = sm_sum;
			}
		);
	}

	template<int d> void VoronoiField<d>::Update_Rho()
	{
		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) { return; }

				int nb_n = nbs_c(cell).size();
				real s = (real)0;
				for (int j = 0; j < nb_n; j++) {
					int nb_p = nbs_c(cell)[j];
					s += pow((real)1 - Softmax(nb_p, cell), (real)beta);
				}
				rho(cell) = (real)(nb_n - c) - s;
			}
		);
	}

	//////////////////////////////////////////////////////////////////////////
	////sensitivities, Fan: subject to change

	template<int d> void VoronoiField<d>::Update_DRho_DX()
	{
		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) { return; }

				int nb_c_n = nbs_c(cell).size();
				real s = (real)0;
				for (int j = 0; j < nb_c_n; j++) {
					int nb_p = nbs_c(cell)[j];
					VectorD drhodx = VectorD::Zero();
					for (int k = 0; k < nb_c_n; k++) {
						int m = nbs_c(cell)[k];
						VectorD dsdx = dS_dX(m, nb_p, cell);
						real S = Softmax(m, cell);
						drhodx += beta * pow(1 - S, beta - 1) * dsdx;
					}
					drho_dx(cell)[j] = drhodx;
				}
			}
		);

		Info("Derivative drho_dx is updated");
	}

	//template<int d> void VoronoiField<d>::Update_DRho_DD()
	//{
	//	int cell_num = grid.Counts().prod();
	//#pragma omp parallel for
	//	for (int i = 0; i < cell_num; i++) {
	//		VectorDi cell = grid.Coord(i);
	//		VectorD pos = grid.Position(cell);
	//		if (active(cell) != (int)TopoCellType::Active)continue;
	//
	//		int nb_c_n = nbs_c[i].size();
	//		for (int j = 0; j < nb_c_n; j++) {
	//			int n = nbs_c[i][j];
	//			MatrixD drhodd = MatrixD::Zero();
	//			for (int k = 0; k < nb_c_n; k++) {
	//				int m = nbs_c[i][k];
	//				MatrixD dsdd = dS_dD(m, n, cell);
	//				real S = Softmax(m, cell);
	//				drhodd -= beta * pow(S, beta - 1) * dsdd;
	//			}
	//
	//			drho_dD[i][j] = drhodd;
	//		}
	//	}
	//}

	//////////////////////////////////////////////////////////////////////////
	////numerical derivatives

	template<int d> void VoronoiField<d>::Numerical_Derivative_DRho_DX()
	{
		real delta_x = (real)1e-6;
		int p_size = particles.Size();

		std::cout << "Numerical DRho_DX" << std::endl;
		Update_DRho_DX();
		Field<real, d> rho_test = rho;
		Field<Array<VectorD>, d> numeric_derv(grid);

		//should not parallelize here
		grid.Iterate_Nodes(
			[&](const VectorDi cell) {
				int nb_c_n = nbs_c(cell).size();
				numeric_derv(cell).resize(nb_c_n);
				for (int j = 0; j < nb_c_n; j++) {
					int nb_p = nbs_c(cell)[j];
					VectorD old_pos = particles.x(nb_p);
					for (int k = 0; k < d; k++) { //iterate through dimensions
						particles.x(nb_p)[k] += delta_x;
						Update_Softmax_Sum();
						Update_Rho();
						numeric_derv(cell)[j][k] = (rho(cell) - rho_test(cell)) / delta_x;
						particles.x(nb_p)[k] = old_pos[k];
					}
					if (!Meso::MathFunc::All_Close(numeric_derv(cell)[j], drho_dx(cell)[j], (real)1e-2, (real)1e-3)) {
						Warn("cell:{}, nb:{}, analytical:{}, numerical:{}", cell, j, drho_dx(cell)[j], numeric_derv(cell)[j]);
					}
				}
			}
		);

		Pass("Finished test of numerical derivative");
	}

	//template<int d> void VoronoiField<d>::Numerical_Derivative_DRho_DD()
	//{
	//	real delta_x = (real)1e-6;
	//	int p_size = particles.Size();
	//
	//	std::cout << "Numerical DRho_DX" << std::endl;
	//	Update_DRho_DD();
	//	Field<real, d> rho_test = rho;
	//	int cell_num = grid.Counts().prod();
	//	Array<Array<MatrixD> > numeric_derv(cell_num);
	//	for (int i = 0; i < cell_num; i++) {
	//		int nb_c_n = nbs_c[i].size();
	//		numeric_derv[i].resize(nb_c_n);
	//		for (int j = 0; j < nb_c_n; j++) {
	//			int nb_p = nbs_c[i][j];
	//			MatrixD old_D = particles.D(nb_p);
	//			for (int k = 0; k < d; k++) {
	//				for (int l = 0; l < d; l++) {
	//					particles.D(nb_p)(k, l) += delta_x;
	//					Update_A();
	//					Update_Softmax_Sum();
	//					Update_Rho();
	//					numeric_derv[i][j](k, l) = (rho.Data()[i] - rho_test.Data()[i]) / delta_x;
	//					particles.D(nb_p)(k, l) = old_D(k, l);
	//				}
	//			}
	//		}
	//	}
	//
	//	for (int i = 0; i < cell_num; i++) {
	//		int nb_c_n = nbs_c[i].size();
	//		for (int j = 0; j < nb_c_n; j++) {
	//			if (!numeric_derv[i][j].isApprox(drho_dD[i][j], 1e-2)) {
	//				std::cout << "cell " << i << ", nb " << j << ",\nanalytical:\n" << drho_dD[i][j]
	//					<< "\nnumerical:\n" << numeric_derv[i][j] << std::endl;
	//			}
	//		}
	//	}
	//}

	template class VoronoiField<2>;
	template class VoronoiField<3>;
}