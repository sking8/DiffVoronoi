#include "hip/hip_runtime.h"
#include "VoronoiField.h"
#include "TopoCellType.h"

namespace Meso {
	//////////////////////////////////////////////////////////////////////////
	////initialization
	template<int d> void VoronoiField<d>::Initialize(const Grid<d> _grid, const Array<VectorD>& points, const int _nb_n, int _beta, real _alpha, real _c)
	{
		int p_num = (int)points.size();
		grid = _grid;
		nb_n = _nb_n;
		particles.Resize(p_num);
		particles.xRef() = points;
		beta = _beta;
		ArrayFunc::Fill(particles.DRef(), _alpha * MatrixD::Identity());
		c = _c;

		////grid attributes
		rho.Init(grid);
		rho.Fill((real)0);
		soft_max_sum.Init(grid);
		soft_max_sum.Fill((real)0);
		active.Init(grid); //Fan: need to consider this, initialize outside of the class
		active.Fill((int)TopoCellType::Active);

		nbs_c.Init(grid);
		drho_dx.Init(grid);
	}

	//////////////////////////////////////////////////////////////////////////
	////field updates

	template<int d> void VoronoiField<d>::Advance(DriverMetaData& metadata)
	{
		Update_A();
		Update_Neighbors();
		Update_Softmax_Sum();
		Update_Rho();
		//Numerical_Derivative_DRho_DX();
	}

	template<int d> void VoronoiField<d>::Update_A()
	{
		particles.Exec_Points(
			[&](const int idx) {
				particles.A(idx) = particles.D(idx) * particles.D(idx).transpose() + epsi_A * MatrixD::Identity();
				/*Info("p: {}", idx);
				std::cout << "A: " << particles.A(idx) << std::endl;
				std::cout << "D: " << particles.D(idx) << std::endl;*/
			}
		);
	}

	template<int d> void VoronoiField<d>::Update_Neighbors()
	{
		////nb particles of each cell
		particles.Update_Searcher();

		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				nbs_c(cell).clear();
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) { return; }

				particles.nbs_searcher.Find_K_Nearest_Nbs(pos, nb_n, nbs_c(cell));
				int nb_n = nbs_c(cell).size();
				if (nb_n > 0) {
					drho_dx(cell).resize(nb_n);
					//drho_dD[idx].resize(nb_c_n);
				}
			}
		);
	}

	template<int d> void VoronoiField<d>::Update_Softmax_Sum()
	{
		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) {
					soft_max_sum(cell) = (real)0.; return;
				}

				real sm_sum = (real)0;
				int nb_c_n = nbs_c(cell).size();
				for (int j = 0; j < nb_c_n; j++) {
					int pid = nbs_c(cell)[j];
					real dis = Dist(pid, pos);
					sm_sum += exp(-dis);
				}
				soft_max_sum(cell) = sm_sum;
			}
		);
	}

	template<int d> void VoronoiField<d>::Update_Rho()
	{
		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) { return; }

				int nb_n = nbs_c(cell).size();
				real s = (real)0;
				for (int j = 0; j < nb_n; j++) {
					int nb_p = nbs_c(cell)[j];
					s += pow((real)1 - Softmax(nb_p, cell), (real)beta);
				}
				rho(cell) = (real)(nb_n - c) - s;
			}
		);
	}

	//////////////////////////////////////////////////////////////////////////
	////sensitivities

	template<int d> void VoronoiField<d>::Update_DRho_DX()
	{
		grid.Exec_Nodes(
			[&](const VectorDi cell) {
				VectorD pos = grid.Position(cell);
				if (active(cell) != (int)TopoCellType::Active) { return; }

				int nb_c_n = nbs_c(cell).size();
				real s = (real)0;
				for (int j = 0; j < nb_c_n; j++) {
					int nb_p = nbs_c(cell)[j];
					VectorD drhodx = VectorD::Zero();
					for (int k = 0; k < nb_c_n; k++) {
						int m = nbs_c(cell)[k];
						VectorD dsdx = dS_dX(m, nb_p, cell);
						real S = Softmax(m, cell);
						drhodx += beta * pow(1 - S, beta - 1) * dsdx;
					}
					drho_dx(cell)[j] = drhodx;
				}
			}
		);

		Info("Derivative drho_dx is updated");
	}

	//////////////////////////////////////////////////////////////////////////
	////numerical derivatives

	template<int d> void VoronoiField<d>::Numerical_Derivative_DRho_DX()
	{
		real delta_x = (real)1e-6;
		int p_size = particles.Size();

		std::cout << "Numerical DRho_DX" << std::endl;
		Update_DRho_DX();
		Field<real, d> rho_test = rho;
		Field<Array<VectorD>, d> numeric_derv(grid);

		//should not parallelize here
		grid.Iterate_Nodes(
			[&](const VectorDi cell) {
				int nb_c_n = nbs_c(cell).size();
				numeric_derv(cell).resize(nb_c_n);
				for (int j = 0; j < nb_c_n; j++) {
					int nb_p = nbs_c(cell)[j];
					VectorD old_pos = particles.x(nb_p);
					for (int k = 0; k < d; k++) { //iterate through dimensions
						particles.x(nb_p)[k] += delta_x;
						Update_Softmax_Sum();
						Update_Rho();
						numeric_derv(cell)[j][k] = (rho(cell) - rho_test(cell)) / delta_x;
						particles.x(nb_p)[k] = old_pos[k];
					}
					if (!Meso::MathFunc::All_Close(numeric_derv(cell)[j], drho_dx(cell)[j], (real)1e-2, (real)1e-3)) {
						Warn("cell:{}, nb:{}, analytical:{}, numerical:{}", cell, j, drho_dx(cell)[j], numeric_derv(cell)[j]);
					}
				}
			}
		);

		Pass("Finished test of numerical derivative");
	}

	template class VoronoiField<2>;
	template class VoronoiField<3>;
}