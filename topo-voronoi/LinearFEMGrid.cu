#include "hip/hip_runtime.h"
#include "LinearFEMGrid.h"
#include "ColorGrid.h"

#include <amgcl/backend/eigen.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/relaxation/spai0.hpp>
#include <amgcl/solver/cg.hpp>
using namespace Meso;

//=================================Helper function================================
template<int d>
Vector<int,d> Corner_Offset(const Vector<int, d>& center, int i) {
	Assert(i < pow(2, d), "Corner_Offset: index out of  range");
	if constexpr (d == 2) { return center + Vector2i(i & 0x1, (i >> 1) & 0x1); }
	else if constexpr (d == 3) { return center + Vector3i(i & 0x1, (i >> 1) & 0x1, (i >> 2) & 0x1); }
	else { Error("Corner_Offset: dimension not supported"); return Vector<int, d>(); }
}

template<int d> void Vector_To_Field(const VectorX& v, Field<Vector<real,d>,d>& f) {
	Typedef_VectorD(d);
	Assert(f.grid.Counts().prod() * d == v.size(), "Vector_To_Field: vector and field should have the same size");
	f.Exec_Nodes(
		[&](const VectorDi node) {
			int idx = f.grid.Index(node);
			f(node) = v.segment<d>(d * idx);
		}
	);
}

//================================Main body=======================================
template<int d> void LinearFEMGrid<d>::Output(DriverMetaData& metadata) {
	//output the displacement field
	Field<Vector<real, d>, d> u_field;
	u_field.Init(grid);
	Vector_To_Field<d>(u, u_field);
	std::string vts_name = fmt::format("u_vts{:04d}.vts", metadata.current_frame);
	bf::path vtk_path = metadata.base_path / bf::path(vts_name);
	VTKFunc::Write_Vector_Field(u_field, vtk_path.string());
}

template<int d> void LinearFEMGrid<d>::Initialize(const Grid<d> _grid, const BoundaryConditionGrid<d>& _bc, const Array<std::tuple<real,real>>& _materials, const Field<short, d>& _material_id) //this is a corner grid
{
	//colored_cell_ptr.resize(Pow(2, d) + 1);
	//for (int i = 0; i < colored_cell_ptr.size(); i++)colored_cell_ptr[i] = i;

	grid = _grid;
	bc = _bc;
	material_id = _material_id;
	for (int i = 0; i < _materials.size(); i++) { Add_Material(_materials[i]); }
	int n = grid.Counts().prod() * d;								// be careful about the difference between cell counts and node counts
	K.resize(n, n); u.resize(n); u.fill((real)0); f.resize(n); f.fill((real)0);
	Allocate_K();
}

template<int d> void LinearFEMGrid<d>::Add_Material(const std::tuple<real, real> material)
{
	auto [youngs, poisson] = material;
	MatrixX Ke0; //stiffness matrix of the material
	LinearFEMFunc::Cell_Stiffness_Matrix<d>(youngs, poisson, grid.dx, Ke0);
	Ke.push_back(Ke0);
}

template<int d> void LinearFEMGrid<d>::Allocate_K()
{
	std::vector<Triplet<real>> elements; //Fan: Can only use std array 
	
	//need to check which side that data resides
	grid.Iterate_Nodes(
		[&](const VectorDi node) {
		int r = grid.Index(node);		//index here is the index in memory
		for (int nb_r = 0; nb_r < grid.Neighbor_Node_Number(); nb_r++) {
			int c = grid.Index(grid.Neighbor_Ring_Node(node, nb_r));
			if (!grid.Valid(c)) { continue; }
			for (int rr = r * d; rr < (r + 1) * d; rr++)for (int cc = c * d; cc < (c + 1) * d; cc++) { elements.push_back(Triplet<real>(rr, cc, (real)0)); }
		}
		}
	);

	K.setFromTriplets(elements.begin(), elements.end());
	K.makeCompressed();

	//ColorGrid::Color<d>(grid.Counts(), colored_cell_ptr, colored_cell_indices);
}

template<int d> void LinearFEMGrid<d>::Update_K_And_f()
{
	////Update K
//	int color_n = colored_cell_ptr.size()-1;
//	for (int c = 0; c < color_n; c++) {	//Fan: this can be improved as in PoissonFunc.h
//#pragma omp parallel for
//		for (int i = colored_cell_ptr[c]; i < colored_cell_ptr[c + 1]; i++) {
//			VectorDi cell = grid.Cell_Grid().Coord(colored_cell_indices[i]); //Fan: this need to be changed, the colored cell indices are not indices in memory
//			int mat_id = material_id(cell);
//			Array<int> corners(pow(2, d), 0);
//			for (int j = 0; j < corners.size(); j++) {
//				corners[j] = grid.Index(Corner_Offset<d>(cell, j));
//			}
//			LinearFEMFunc::Add_Cell_Stiffness_Matrix<d>(K, Ke[mat_id], corners);
//		}
//	}

	//Fan: not parallelized version, to be changed
	grid.Cell_Grid().Iterate_Nodes(
		[&](const VectorDi node) {
			Array<int> corners(pow(2, d), 0);
			for (int j = 0; j < corners.size(); j++) {
				corners[j] = grid.Index(Corner_Offset<d>(node, j));
			}
			int mat_id = material_id(node);
			LinearFEMFunc::Add_Cell_Stiffness_Matrix<d>(K, Ke[mat_id], corners); //Fan: exception here, may be some indexing issue!
		}
	);

	////Update rhs
	f.fill((real)0);
	for (auto& b : bc.forces) {
		VectorDi node = b.first; VectorD force = b.second;
		for (int axis = 0; axis < d; axis++) { int idx = grid.Index(node) * d + axis; f[idx] += force[axis]; } //Fan: Is the index usage right here. Yes
	}

	////Update bc
	for (auto& b : bc.psi_D_values) {
		VectorDi node = b.first; VectorD dis = b.second;
		for (int axis = 0; axis < d; axis++) {
			int idx = grid.Index(node) * d + axis;			//Fan: is the index usage right here. Yes
			LinearFEMFunc::Set_Dirichlet_Boundary_Helper(K, f, idx, dis[axis]);
		}
	}
}

template<int d> void LinearFEMGrid<d>::Solve()
{
	u.fill((real)0);

//	multigrid_params.use_auto_calculated_levels = true;
//	multigrid_params.dof_on_cell = false;
//	multigrid_params.block_size = d;
//	multigrid_params.use_gpu = true;
//	multigrid_params.init_hier_on_gpu = true;	////calculate hier on CPU to avoid GPU memory crash
//
//#ifdef USE_CUDA
//	if (multigrid_params.use_gpu) {
//		//GeometricMultiGrid::GMGPCG_GPU<d>(K,u,f,grid.node_counts,multigrid_params);
//		gmg_solver_gpu.update_A_levels = true;
//
//		gmg_solver_gpu.Initialize(K, grid.node_counts, multigrid_params, &material_id);
//		gmg_solver_gpu.Solve(u, f);
//	}
//	else {
//		gmg_solver_cpu.update_A_levels = true;
//		gmg_solver_cpu.Initialize(K, grid.node_counts, multigrid_params, &material_id);
//		gmg_solver_cpu.Solve(u, f);
//	}
//#else
//	gmg_solver_cpu.Initialize(K, grid.node_counts, multigrid_params, &material_id);
//	gmg_solver_cpu.Solve(u, f);
//#endif

	// use amgcl solver for solving the system
	// Setup the solver:
	typedef amgcl::make_solver<
		amgcl::amg<
		amgcl::backend::eigen<real>,
		amgcl::coarsening::smoothed_aggregation,
		amgcl::relaxation::spai0
		>,
		amgcl::solver::cg<amgcl::backend::eigen<real> >
	> Solver;

	Solver solve(K);
	std::cout << solve << std::endl;

	// Solve the system for the given RHS:
	auto [a, error]= solve(f, u);
	Info("Solver finished within {} iters with error {}.", a, error);
}

template<int d> void LinearFEMGrid<d>::Compute_Cell_Displacement(const VectorX& u, const VectorDi& cell, VectorX& cell_u) const
{
	int number_of_cell_nodes = pow(2, d);
	cell_u.resize(number_of_cell_nodes * d);
	for (int i = 0; i < number_of_cell_nodes; i++) {
		VectorDi nb_node = Corner_Offset<d>(cell, i); int nb_node_mtx_idx = grid.Index(nb_node);
		for (int j = 0; j < d; j++)cell_u(i * d + j) = u(nb_node_mtx_idx * d + j);
	}
}

template<int d> void LinearFEMGrid<d>::Compute_Elastic_Energy(Field<real, d>& energy) const
{
	energy.Init(grid.Cell_Grid().Counts(), (real)0);

	Grid<d> cell_grid = grid.Cell_Grid();
	cell_grid.Exec_Nodes(
		[&](const VectorDi cell) {
			int mat_id = material_id(cell); if (mat_id == -1)return;
			VectorX cell_u; Compute_Cell_Displacement(u, cell, cell_u);
			const MatrixX& K0 = Ke[mat_id];
			energy(cell) = (real)0.5*cell_u.dot(K0 * cell_u);
		}
	);
}

template class LinearFEMGrid<2>;
template class LinearFEMGrid<3>;
