#include "hip/hip_runtime.h"
#include "LinearFEMGrid.h"
#include <amgcl/backend/eigen.hpp>
#include <amgcl/make_solver.hpp>
#include <amgcl/amg.hpp>
#include <amgcl/coarsening/smoothed_aggregation.hpp>
#include <amgcl/relaxation/spai0.hpp>
#include <amgcl/solver/cg.hpp>
using namespace Meso;

template<int d> int LinearFEMGrid<d>::Compact_Idx(int idx) {
	return compact_indices[idx];
}

//================================Main body=======================================
template<int d> void LinearFEMGrid<d>::Output(DriverMetaData& metadata) {
	//output the displacement field
	std::string vts_name = fmt::format("u_vts{:04d}.vts", metadata.current_frame);
	bf::path vtk_path = metadata.base_path / bf::path(vts_name);
	VTKFunc::Write_Vector_Field(u, vtk_path.string());
	VTKFunc::Write_Boundary_Condition(bc, grid, metadata.base_path);
}

template<int d> void LinearFEMGrid<d>::Initialize(const Grid<d> _grid, const BoundaryConditionGrid<d>& _bc, const Array<std::tuple<real,real>>& _materials, const Field<short, d>& _material_id) //this is a corner grid
{
	grid = _grid;
	bc = _bc;
	material_id = _material_id;
	for (int i = 0; i < _materials.size(); i++) { Add_Material(_materials[i]); }

	u.Init(grid, VectorD::Zero());
	f.Init(grid, VectorD::Zero());
	int n = grid.Counts().prod() * d;	// be careful about the difference between cell counts and node counts
	compact_indices.resize(grid.Memory_Size(), (int)-1);
	
	K.resize(n, n);

	//create the mapping
	int counter = 0;
	grid.Iterate_Nodes(
		[&](const VectorDi node) {
			if (grid.Valid(node)) { 
				compact_indices[grid.Index(node)] = counter; 
				counter++;
			}
		}
	);
	Allocate_K();
}

template<int d> void LinearFEMGrid<d>::Add_Material(const std::tuple<real, real> material)
{
	auto [youngs, poisson] = material;
	MatrixX Ke0; //stiffness matrix of the material
	LinearFEMFunc::Cell_Stiffness_Matrix<d>(youngs, poisson, grid.dx, Ke0);
	Ke.push_back(Ke0);
}

template<int d> void LinearFEMGrid<d>::Allocate_K()
{
	std::vector<Triplet<real>> elements; //Fan: Can only use std array 
	
	//need to check which side that data resides
	grid.Iterate_Nodes(
		[&](const VectorDi node) {
		int r = Compact_Idx(grid.Index(node));
		for (int nb_r = 0; nb_r < grid.Neighbor_Ring_Number(); nb_r++) {
			VectorDi ring_node = Grid<d>::Neighbor_Ring_Node(node, nb_r);
			int c = grid.Index(ring_node);
			if (!grid.Valid(ring_node)) { continue; }
			c = Compact_Idx(c);
			for (int rr = r * d; rr < (r + 1) * d; rr++) {
				for (int cc = c * d; cc < (c + 1) * d; cc++) { 
					elements.push_back(Triplet<real>(rr, cc, (real)0)); 
				}
			}
		}
		}
	);

	K.setFromTriplets(elements.begin(), elements.end());
	K.makeCompressed();
}

template<int d> void LinearFEMGrid<d>::Update_K()
{
	//Update K
	//Fan: not parallelized version, to be changed
	grid.Cell_Grid().Iterate_Nodes(
		[&](const VectorDi node) {
			Array<int> corners(pow(2, d), 0);
			for (int j = 0; j < corners.size(); j++) {
				corners[j] = Compact_Idx(grid.Index(LinearFEMFunc::Corner_Offset<d>(node, j)));
			}
			int mat_id = material_id(node);
			LinearFEMFunc::Add_Cell_Stiffness_Matrix<d>(K, Ke[mat_id], corners);
		}
	);
}

template<int d> void LinearFEMGrid<d>::Solve()
{
	//VectorX f_v, u_v;
	VectorX f_v(K.cols());
	VectorX u_v(K.cols());
	f_v.fill((real)0);
	u_v.fill((real)0);

	grid.Exec_Nodes(
		[&](const VectorDi node) {
			int idx=Compact_Idx(grid.Index(node));
			if (idx) { f_v.segment<d>(idx * d) = f(node); }
		}
	);

	////Update rhs
	for (auto& b : bc.forces) {
		VectorDi node = b.first; VectorD force = b.second;
		int idx = Compact_Idx(grid.Index(node));
		f_v.segment<d>(idx*d) = force;
	}

	//Update bc
	for (auto& b : bc.psi_D_values) {
		VectorDi node = b.first; VectorD dis = b.second;
		for (int axis = 0; axis < d; axis++) {
			int idx = Compact_Idx(grid.Index(node)) * d + axis;		//need to be careful here when the grid has padding
			LinearFEMFunc::Set_Dirichlet_Boundary_Helper(K, f_v, idx, dis[axis]);
		}
	}
	
	// use amgcl solver for solving the system
	// Setup the solver:
	typedef amgcl::make_solver<
		amgcl::amg<
		amgcl::backend::eigen<real>,
		amgcl::coarsening::smoothed_aggregation,
		amgcl::relaxation::spai0
		>,
		amgcl::solver::cg<amgcl::backend::eigen<real> >
	> Solver;

	Solver solve(K); //K is singular in 3D
	std::cout << solve << std::endl;

	// Solve the system for the given RHS:
	auto [iter, error]= solve(f_v, u_v);
	Info("AMGCL solver with Eigen backend finished within {} iters with error {}.", iter, error);

	grid.Exec_Nodes(
		[&](const VectorDi node) {
			int idx = Compact_Idx(grid.Index(node));
			if (idx) { u(node) = u_v.segment<d>(idx * d); }
		}
	);
}

template<int d> void LinearFEMGrid<d>::Compute_Cell_Displacement(const VectorDi& cell, VectorX& cell_u) const
{
	int number_of_cell_nodes = pow(2, d);
	cell_u.resize(number_of_cell_nodes * d);
	for (int i = 0; i < number_of_cell_nodes; i++) {
		VectorDi nb_node = LinearFEMFunc::Corner_Offset<d>(cell, i);
		cell_u.segment<d>(i*d) = u(nb_node);
	}
}

template<int d> void LinearFEMGrid<d>::Compute_Elastic_Energy(Field<real, d>& energy) const
{
	energy.Init(grid.Cell_Grid().Counts(), (real)0);

	Grid<d> cell_grid = grid.Cell_Grid();
	cell_grid.Exec_Nodes(
		[&](const VectorDi cell) {
			int mat_id = material_id(cell); if (mat_id == -1)return;
			VectorX cell_u; Compute_Cell_Displacement(cell, cell_u);
			const MatrixX& K0 = Ke[mat_id];
			energy(cell) = (real)0.5*cell_u.dot(K0 * cell_u);
		}
	);
}

template class LinearFEMGrid<2>;
template class LinearFEMGrid<3>;
