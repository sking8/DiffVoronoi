//////////////////////////////////////////////////////////////////////////
// Auxiliary Function CUDA
// Copyright (c) (2018-), Bo Zhu
// This file is part of SimpleX, whose distribution is governed by the LICENSE file.
//////////////////////////////////////////////////////////////////////////

#include "AmgclSolverGPU.h"

//////////////////////////////////////////////////////////////////////////
//// single function API
////gpu amgcl
template<int d> bool AMGCL_GPU(const SparseMatrix<real>& A,VectorN<real>& x,const VectorN<real>& b,const AmgclSolver::Params params/*=Params()*/,bool verbose/*=false*/)
{
#ifndef USE_CUDA
	std::cerr<<"Error: [AMGCL_GPU] USE_CUDA disabled"<<std::endl;
	return false;
#else
    // ref to https://buildmedia.readthedocs.org/media/pdf/amgcl/latest/amgcl.pdf, Listing 2.6
	Timer timer;timer.Reset();
    typedef amgcl::make_solver<
        amgcl::amg<
            amgcl::backend::cuda<real>,
            amgcl::coarsening::smoothed_aggregation,
            amgcl::relaxation::spai0
            >,
        amgcl::solver::bicgstab<
            amgcl::backend::cuda<real>
            >
        > CudaSolver;
    
    amgcl::backend::cuda<real>::params bprm;
    hipsparseCreate(&bprm.cusparse_handle);

	if(!Is_Cuda_Context_Initialized())Initialize_Cuda_Context();///one bug

	int n=A.rows();
    thrust::device_vector<real> b_dev(b.data(), b.data() + n);
    thrust::device_vector<real> x_dev(x.data(), x.data() + n);
	if(verbose){timer.Elapse_And_Output_And_Reset("GPU AMGCL: cpu to gpu transfer");}

    CudaSolver::params prm;
    prm.solver.maxiter=params.max_iter_num;
    prm.solver.tol=params.tolerance;
    prm.solver.ns_search=params.ns_search;

    CudaSolver solve(A, prm, bprm); // Eigen::SparseMatrix is used directly
	if(verbose){timer.Elapse_And_Output_And_Reset("GPU AMGCL: solver setup");}

    int iters;
    double error;
    std::tie(iters, error) = solve(b_dev, x_dev);
	if(verbose)timer.Elapse_And_Output_And_Reset("GPU AMGCL: gpu solving");
    if(verbose)std::cout <<"#Iters="<< iters << ", error=" << error << std::endl;

    thrust::copy(x_dev.begin(),x_dev.end(),x.data());
	if(verbose){timer.Elapse_And_Output_And_Reset("GPU AMGCL: gpu to cpu transfer");AuxFunc::Seperation_Line();}

	return true;
#endif
}


template bool AMGCL_GPU<2>(const SparseMatrix<real>& A,VectorN<real>& x,const VectorN<real>& b,const AmgclSolver::Params params/*=Params()*/,bool verbose/*=false*/);
template bool AMGCL_GPU<3>(const SparseMatrix<real>& A,VectorN<real>& x,const VectorN<real>& b,const AmgclSolver::Params params/*=Params()*/,bool verbose/*=false*/);